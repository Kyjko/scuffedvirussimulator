#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <memory.h>
#include <stdlib.h>
#include <SDL.h>

#undef main

#define W 1024
#define H 1024

#define N 65000
#define MIN_INFECTION_RANGE 30.0f

short quit = 0;

static SDL_Window* w;
static SDL_Renderer* r;
static SDL_Texture* txt;
static unsigned int* people;
static unsigned int* d_people;

static unsigned int* people_pixels;
static unsigned int* d_people_pixels;

static unsigned int global_framecount = 0;

__global__ void update_people(unsigned int* d_people, int n) {
    unsigned long idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx > n)
        return;
    
    unsigned int sp = d_people[idx];
    for(int i = 0; i < n; i++) {
        unsigned int p = d_people[i];
        int x = (p>>8)&2047; 
        int y = p>>19;    
        int sx = (sp>>8)&2047; 
        int sy = sp>>19;
        if(x != sx && y != sy) {
            float dist2 = (x-sx)*(x-sx) + (y-sy)*(y-sy);
            if(dist2 <= MIN_INFECTION_RANGE && p&1 && p&(1<<1)) {
                d_people[idx] |= 1;
            } 
        }
    }
}

__global__ void update_people_age(unsigned int* d_people, int n, int d_age) {
    unsigned long idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx > n)
        return;

    if(d_people[idx]&(1<<1)) {
        d_people[idx] += ((d_people[idx]&1 ? 10 : 1)*d_age)<<2;
    }
    if((d_people[idx]&252)>>2 >= 63) {
        d_people[idx] &= 1073741820;
    }

}

__global__ void update_people_pixels(unsigned int* d_people, unsigned int* d_people_pixels, int n) {
    unsigned long idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx > n)
        return;

    
    d_people_pixels[idx] = (((d_people[idx]&1) ? 255/(((d_people[idx]&252)>>7)+1) : 0)*(d_people[idx]&(1<<1))>>1)<<24 |
                           (((d_people[idx]&1) ? 0 : 255/(((d_people[idx]&252)>>7)+1))*(d_people[idx]&(1<<1))>>1)<<16 |
                           (255-((d_people[idx]&252)>>2)*(d_people[idx]&(1<<1))>>1)<<8 |
                           255;
}

errno_t init_sdl() {
    if(SDL_Init(SDL_INIT_VIDEO) < 0)
        return -1;
    
    w = SDL_CreateWindow("Numpu", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, W, H, SDL_WINDOW_SHOWN);
    if(w == NULL)
        return -2;
    r = SDL_CreateRenderer(w, -1, SDL_RENDERER_ACCELERATED);
    if(r == NULL)
        return -3;

    txt = SDL_CreateTexture(r, SDL_PIXELFORMAT_RGBA8888, SDL_TEXTUREACCESS_STREAMING, W, H);
    if(txt == NULL)
        return -4;
    
    return 0;

}

void render() {
    
    update_people<<< 1024, 1024 >>>(d_people, N);
    update_people_age<<< 1024, 1024 >>>(d_people, N, (global_framecount%5) ? 0 : 1);
    //update_people_pixels<<< 1024, 1024 >>>(d_people, d_people_pixels, N);

    hipMemcpy(people, d_people, sizeof(unsigned int)*N, hipMemcpyDeviceToHost);
    //hipMemcpy(people_pixels, d_people_pixels, sizeof(unsigned int)*N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) {
        if((float)(rand()/(float)RAND_MAX) < 0.01f && !(people[i]&1))
            people[i] &= ~2;
        
        unsigned int p = people[i];
        // ......11 -> alive and infected
        // ......01/0 -> dead
        // ......10 -> alive and healthy
        SDL_SetRenderDrawColor(r, ((p&1) ? max(255/(((p&252)>>7)+1), 0) : 0)*(p&(1<<1))>>1, ((p&1) ? 0 : 255/(((p&252)>>7)+1))*(p&(1<<1))>>1, 255-((p&252)>>2)*(p&(1<<1))>>1, 255);
        SDL_Rect rect;
        rect.x = (p>>8)&2047;
        rect.y = p>>19;
        rect.w = 3;
        rect.h = 3;
        SDL_RenderFillRect(r, &rect);
        
        SDL_RenderDrawPoint(r, (p>>8)&2047, p>>19);
        
    }

    //SDL_UpdateTexture(txt, NULL, people_pixels, W);
    //SDL_RenderCopy(r, txt, NULL, NULL);

    global_framecount++;
}

void eventloop() {
    while(quit != 1) {
        SDL_Event e;
        while(SDL_PollEvent(&e) != NULL) {
            switch(e.type) {
                case SDL_QUIT:
                    quit = 1;
                    break;
            }
        }
        SDL_SetRenderDrawColor(r, 0, 0, 0, 0);
        SDL_RenderClear(r);
        render();
        SDL_RenderPresent(r);
    }

    SDL_DestroyRenderer(r);
    SDL_DestroyWindow(w);
    w = NULL;
    r = NULL;
    SDL_Quit();
}

int main(int argc, char** argv) {
    srand((unsigned)time(NULL));

    //init sdl
    if(init_sdl() < 0) {
        perror("cannot initialize SDL!");    
        return -1;
    }

    people = (unsigned int*)malloc(sizeof(unsigned int)*N);
    people_pixels = (unsigned int*)malloc(sizeof(unsigned int)*N);
    for(int i = 0; i < N; i++) {
        // everyone starts from random ages between 1 and 50
        // x and y coordinates are from 0-1920 respectively, they fit in 11 bits
        people[i] = (int)((float)(rand()/(float)RAND_MAX)*W.0f);
        people[i] <<= 11;
        people[i] += (int)((float)(rand()/(float)RAND_MAX)*H.0f);
        people[i] <<= 6;
        people[i] += (int)((float)(rand()/(float)RAND_MAX)*40.0f);
        people[i] <<= 2;
        people[i] += 2;
        //people[i] += 1;
    }
    for(int i = 0; i < 1; i++) {
        people[i] |= 1;
    }
    hipMalloc((void**)&d_people, sizeof(unsigned int)*N);
    hipMalloc((void**)&d_people_pixels, sizeof(unsigned int)*N);
    
    hipMemcpy(d_people, people, sizeof(unsigned int)*N, hipMemcpyHostToDevice);

    eventloop();

    free(people);
    free(people_pixels);
    hipFree(d_people);
    hipFree(d_people_pixels);
    people = NULL;
    d_people = NULL;
    return 0;
}
